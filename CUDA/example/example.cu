/*
*
*      Filename: example.cu
*
*        Author: Haibo Hao
*        Email : haohaibo@ncic.ac.cn
*   Description: ---
*        Create: 2018-04-18 19:16:29
* Last Modified: 2018-04-18 19:16:29
**/

#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void kernel(){
    unsigned long a, b, c;
    a = 24;
    b = 12;
    c = a + b;
}

int main(int argc, char* argv[]){
    hipFree(0);
    kernel<<<1,1>>>();
    hipDeviceSynchronize();
    return  0;
}

